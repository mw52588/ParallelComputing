#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

__global__ void kernel( int **dev_h, size_t pitch1, int **dev_g, size_t pitch2, int N) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	
	int *rowa = (int*) ((char*)dev_h + idx * pitch1);
	int *rowb = (int*) ((char*)dev_g + idx * pitch2);
	
	int gpu_i = idx;
	int gpu_j;
	
   if (idx < N) {
		
   }
 
    __syncthreads();

		for (int iteration = 0; iteration < limit; iteration++) {
		
		int t = N/4;
		h[t][t] = 37;
		h[t][t+1] = 37;
		h[t+1][t] = 37;
		h[t+1][t+1] = 37;

		for (int i = 1; i < N-1; i++) {
			for (int j = 1; j < N-1; j++){ 
				g[i][j] = .25 * (h[i-1][j] + h[i+1][j] + h[i][j-1] + h[i][j+1]);
				h[i][j] = g[i][j];
				h[t][t] = 37;
				h[t][t+1] = 37;
				h[t+1][t] = 37;
				h[t+1][t+1] = 37;	
			}
		}
	
	}
	
}


int main(void) {
	int size = 0;
	int T = 0;
	int B = 0;
	int limit = 1;	
	
	do {
		printf("Enter number for threads per block:(Maximum number of threads per block is 1024)\n");
        scanf(" %d", &T);getchar();
   	}while (T > 1024 || T <= 0);

   	 do {
		printf("Enter blocks per grid: (Maximum number of blocks per grid is 65535)\n");
      	scanf(" %d", &B);getchar();
  	 }while (B > 65535 || B <= 0);
	
	do {
		printf("Enter size of the square room:\n");
      	scanf(" %d", &size);getchar();
  	}while(size <= 0 );

	 const int N = size ; 
	 int block_size = B;
	double h[N][N];
	double g[N][N];
	int **dev_h;
	int **dev_g;
	double s = N/10;
	
	int left = (int)(s * 3);
	int middle = (int)(s * 4) + left;
	int right = (int)s*7 + 1 + left + middle;
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			h[i][j] = 0;
		}
	}

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			h[0][i] = 20;
			h[i][0] = 20;
			h[N-1][i] = 20;
			h[i][N-1] = 20;
		}
	}

	for (int i = 0; i < N; i++) {

		if (i >= left && i < middle) {
			h[0][i] = 100;
		}
		else {
			h[0][i] = 20;
		}		
	}
	size_t pitch1;
	hipMallocPitch(&dev_h, &pitch1, N * sizeof(int), N);
	size_t pitch2;
	hipMallocPitch(&dev_g, &pitch2, N * sizeof(int), N);
	
	hipMemcpy2D(dev_h, pitch1, a, N * sizeof(int), N * sizeof(int),N, hipMemcpyHostToDevice);
    hipMemcpy2D(dev_g, pitch2, b, N * sizeof(int), N * sizeof(int),N, hipMemcpyHostToDevice);
	
	kernal <<< n_blocks, block_size >>> (dev_h, pitch1, dev_g, pitch2, N);
	
	for (int iteration = 0; iteration < limit; iteration++) {
		
		int t = N/4;
		h[t][t] = 37;
		h[t][t+1] = 37;
		h[t+1][t] = 37;
		h[t+1][t+1] = 37;

		for (int i = 1; i < N-1; i++) {
			for (int j = 1; j < N-1; j++){ 
				g[i][j] = .25 * (h[i-1][j] + h[i+1][j] + h[i][j-1] + h[i][j+1]);
				h[i][j] = g[i][j];
				h[t][t] = 37;
				h[t][t+1] = 37;
				h[t+1][t] = 37;
				h[t+1][t+1] = 37;	
			}
		}
	
	}
	

	
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			printf("\nh[%d][%d] = %f",i,j, h[i][j]);
		}
		printf("\n");
	}       

	return 0;
}