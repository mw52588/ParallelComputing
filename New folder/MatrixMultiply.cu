#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

void random_ints(int* a, int N)
{
   int i;
   for (i = 0; i < N; ++i)
    a[i] = rand();
}

void SeqMatMult(int* a, int* b, int* c, int N)
{
	int sum = 0;
	int i, j, k = 0;
	for (i = 0; i < N; ++i) {
		for (j = 0; j < N; ++j) {
			for (k = 0; k < N; ++k) {
				c[i+N*j] += a[i+N*k] * b[k+N*j];
			}
		}
	}
}

__global__ void multiply( int *a, int *b, int *c ) {
	__shared__ int temp[THREADS_PER_BLOCK];
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	temp[threadIdx.x] = a[index] * b[index];
	__syncthreads();
	if( 0 == threadIdx.x ) {
		int sum = 0;
		for( inti= 0; i< THREADS_PER_BLOCK; i++ )
			sum += temp[i];
		atomicAdd( c , sum );
	}
}


int main(void){
    int N = 0; //array Size.
    int T = 0;
    int B = 0;
    bool compareArray = true;

    do {
        printf("Enter number for threads per block:(Maximum number of threads per block is 1024)\n");
        scanf(" %d", &T);getchar();
    }while (T > 1024 || T <= 0);

    do {
        printf("Enter blocks per grid: (Maximum number of blocks per grid is 65535)\n");
        scanf(" %d", &B);getchar();
    }while (B > 65535 || B <= 0);

    do {
        printf("Enter number for size of the array:\n");
        scanf(" %d", &N);getchar();
    }while(N <= 0);

    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;
	int size = N * sizeof( int );
	
	hipMalloc( (void**)&dev_a, size );
	hipMalloc( (void**)&dev_b, size );
	hipMalloc( (void**)&dev_c, sizeof( int ) );
    
	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( sizeof( int ) );
	
	random_ints(a, N);
	random_ints(b, N);
	
	// copy inputs to device
	hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice);

    
    multiply<<<N/T, B>>>(dev_a, dev_b, dev_c, N);

	hipMemcpy(c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	free(a);
	free(b);
	free(c);
		
    return 0;
}








