
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

__global__ void kernel( int **dev_a , size_t pitch1, int **dev_b , size_t pitch2, int **dev_c , size_t pitch3, int N) {
     int idx=threadIdx.x+blockIdx.x*blockDim.x;
      int idy=threadIdx.y+blockIdx.y*blockDim.y;


     //Pitch Slices per Row for 2D Array
     int* rowa = (int*)((char*)dev_a + idx*pitch1);
     int* rowb = (int*)((char*)dev_b + idx*pitch2);
     int* rowc = (int*)((char*)dev_c + idx*pitch3);
       
      int gpu_i =idx;
      int gpu_j;
           for(gpu_j = 0;gpu_j< N;gpu_j++)
            {
                int sum=0;
              for (int gpu_k = 0; gpu_k < N; gpu_k++) {
                    int* rowb = (int*)((char*)dev_b + gpu_k*pitch2);
                  sum+= rowa[gpu_k] * rowb[gpu_i];
              }
              rowc[gpu_j] =sum;
           }
 
    __syncthreads();
   
}

int main(int argc, char *argv[]) {
        int i;
        int j;
        int k;
	int size = 0;
	int B = 0;
	int T = 0;
        //Kernel Variables
       

        //CUDA GRID BLOCK SIZE AND NUMBER OF BLOCKS

	  do {
      	 	 printf("Enter number for threads per block:(Maximum number of threads per block is 1024)\n");
        	scanf(" %d", &T);getchar();
   	 }while (T > 1024 || T <= 0);

   	 do {
     	   printf("Enter blocks per grid: (Maximum number of blocks per grid is 65535)\n");
      	  scanf(" %d", &B);getchar();
  	  }while (B > 65535 || B <= 0);

   	 do {
     	   printf("Enter number for size of the array:\n");
      	  scanf(" %d", &size);getchar();
  	  }while(size <= 0 || size > B * T);
	


	
        int block_size = B;
        const int N = size ;  // Number of elements in arrays


        int n_blocks = N/block_size + (N%block_size == 0 ? 0:1);
	 int a[N][N];
	int **dev_a;
        int b[N][N];
	int **dev_b;
        int c[N][N];
	int **dev_c;

		
	 for (i = 0; i < N; i++) {
             for (j = 0; j < N; j++) {
                  a[i][j] = 1;
                  b[i][j] = j;
                  c[i][j] = 0;
             }
        }
        printf("\n A = ");
        for (i = 0; i < N; i++) {
             printf("\n");
             for (j = 0; j < N; j++) {
                  printf(" %d ",a[i][j]);
             }
        }
        printf("\n B = ");
        for (i = 0; i < N; i++) {
             printf("\n");
             for (j = 0; j < N; j++) {
                  printf(" %d ",b[i][j]);
             }
        }

        // Memory Allocation
        size_t pitch1;
         hipMallocPitch(&dev_a, &pitch1, N * sizeof(int), N);
         // Allocate 2Darray on device
        size_t pitch2;
         hipMallocPitch(&dev_b, &pitch2, N * sizeof(int), N);
         // Allocate 2Darray on device
        size_t pitch3;
         hipMallocPitch(&dev_c, &pitch3, N * sizeof(int), N);
         // Allocate 2Darray on device

	hipEvent_t start, end, s, e;                    // using cuda events to measure time
    	float time , t;                               // which is applicable for asynchronous code also

    	hipEventCreate(&start);                 // instrument code to measure start time
    	hipEventCreate(&end);
	hipEventCreate(&s);                 // instrument code to measure start time
    	hipEventCreate(&e);
	
        // Copy Data to device from host
        hipMemcpy2D(dev_a, pitch1, a, N * sizeof(int), N * sizeof(int),N, hipMemcpyHostToDevice);
        hipMemcpy2D(dev_b, pitch2, b, N * sizeof(int), N * sizeof(int),N, hipMemcpyHostToDevice);
        hipMemcpy2D(dev_c, pitch3, c, N * sizeof(int), N * sizeof(int),N, hipMemcpyHostToDevice);

        // call kernel
	 hipEventRecord(start, 0 );
        kernel <<< n_blocks, block_size >>>( dev_a,pitch1, dev_b,pitch2, dev_c,pitch3, N);
	 hipEventRecord(end, 0 );                 // instrument code to measure end time
   	 hipEventSynchronize(end);
    	 hipEventElapsedTime(&time, start, end);

        // Retrieve result from device and store it in host array
        hipMemcpy2D(a,N * sizeof(int), dev_a,pitch1,N * sizeof(int),N, hipMemcpyDeviceToHost);
        hipMemcpy2D(b,N * sizeof(int), dev_b,pitch2,N * sizeof(int),N, hipMemcpyDeviceToHost);
        hipMemcpy2D(c,N * sizeof(int), dev_c,pitch3,N * sizeof(int),N, hipMemcpyDeviceToHost);

        


        printf("\n C = ");
        for (i = 0; i < N; i++) {
             printf("\n");
             for (j = 0; j < N; j++) {
                  printf(" %d ",c[j][i]);
             }
        }
	hipEventRecord(s, 0);
	 for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			int sum = 0;
			for (int k = 0; k < N; k++) {
				sum += a[i][k] * b[k][j];
			}
		c[i][j] = sum;
		}
	}
	hipEventRecord(e, 0 );                 // instrument code to measure end time
    	hipEventSynchronize(e);
    	hipEventElapsedTime(&t, s, e);


	  printf("\n C = ");
        for (i = 0; i < N; i++) {
             printf("\n");
             for (j = 0; j < N; j++) {
                  printf(" %d ",c[i][j]);
             }
        }


	printf("\n\nGPU Time using CUDA events: %f ms\n", time);
    	printf("CPU Time using CUDA events: %f ms\n", t);
    	hipEventDestroy(start);
    	hipEventDestroy(end);
	hipEventDestroy(s);
	hipEventDestroy(e);
	// Free GPU Variables
        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_c);
	
       return 0;

}
