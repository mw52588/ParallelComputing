#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>

__global__ void add(int *a,int *b, int *c, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < N) {
        c[tid] = a[tid]+b[tid];
    }
}

void addHost(int *a, int *b, int *c, int N) {
    for(int i =0; i < N; i++) {
        c[i] = a[i] + b[i];
    }
}

int main(void){
    static int N = 0; //array Size.
    static int T = 0;
    static int B = 0;
    bool compareArray = true;

    do {
        printf("Enter number for threads per block:(Maximum number of threads per block is 1024)\n");
        scanf(" %d", &T);getchar();
    }while (T > 1024 || T <= 0);

    do {
        printf("Enter blocks per grid: (Maximum number of blocks per grid is 65535)\n");
        scanf(" %d", &B);getchar();
    }while (B > 65535 || B <= 0);

    do {
        printf("Enter number for size of the array:\n");
        scanf(" %d", &N);getchar();
    }while(N <= 0);

    int a[N],b[N],c[N];
    int hostc[N];
    int *dev_a, *dev_b, *dev_c;
    hipMalloc((void**)&dev_a,N * sizeof(int));
    hipMalloc((void**)&dev_b,N * sizeof(int));
    hipMalloc((void**)&dev_c,N * sizeof(int));
   
    for(int i=0;i<N;i++) {
        a[i] = i;
        b[i] = i;
    }

    hipEvent_t start, end, s, e;                    // using cuda events to measure time
    float time , t;                               // which is applicable for asynchronous code also

    hipEventCreate(&start);                 // instrument code to measure start time
    hipEventCreate(&end);
   
    hipMemcpy(dev_a, a , N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b , N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c , N*sizeof(int),hipMemcpyHostToDevice);
	hipEventRecord(start, 0 );
    add<<<B,T>>>(dev_a,dev_b,dev_c, N);
	hipEventRecord(end, 0 );                 // instrument code to measure end time
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);
	
    printf("Host to Device: \n");
    for(int i=0;i<N;i++) {
        printf("%d+%d=%d\n",a[i],b[i],c[i]);
    }
   

    printf("\n\nHost process\n");
   
    hipEventCreate(&s);                 // instrument code to measure start time
    hipEventCreate(&e);
    hipEventRecord(s, 0 );
    addHost(a, b, hostc, N);
    hipEventRecord(e, 0);
    hipEventSynchronize(e);
    hipEventElapsedTime(&t, s, e);

   
   
    for (int i =0; i < N; i++) {
        printf("%d+%d=%d\n",a[i],b[i],hostc[i]);
    }
    printf("\n\nCheck to see if device and host are equal\n");
    for (int i = 0; i < N; i++) {
        if (hostc[i] != c[i]) {
            printf("Host c is not equal to device c therefore host and device are not equal");
            compareArray = false;
            break;
        }
        printf("hostc[%i] = %i AND devicec[%i] = %i\n",i,c[i],i,hostc[i]);
    }

    if (compareArray == true) {
   
        printf("\nHostC and DeviceC are the same\n");
    }
    else {
        printf("\nHostC and DeviceC are not the same\n");
    }

    printf("GPU Time using CUDA events: %f ms\n", time);
    printf("CPU Time using CUDA events: %f ms\n", t);
    hipEventDestroy(start);
    hipEventDestroy(end);
	hipEventDestroy(s);
	hipEventDestroy(e);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
   
    return 0;

}